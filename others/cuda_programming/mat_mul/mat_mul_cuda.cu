/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

 // System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv, int block_size, const dim3 &dimsA, const dim3 &dimsB) {
    // allocate host memory for matrix A, B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));
    hipStream_t stream;

    // initialize host memory
    const float valA = 1.0f;
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, valA);
    ConstantInit(h_B, size_B, valB);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));
    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }
    // check mem allocation
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // allocate cuda events that we will use for timing
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // copy host mem to device
    checkCudaErrors(hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

}


/**
 * Program main
 */
int main(int argc, char **argv) {
    printf("[Matrix Multiply Using CUDA] - Starting...\n");
    if (checkCmdLineFlag(argc, (const char **)argv, "help") || checkCmdLineFlag(argc, (const char **)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("\t-wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("\t-wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("Note: Outer matrix dimensions of A & B matrices must be equal.\n");
        exit(EXIT_SUCCESS);
    }

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char **)argv);

    // declare the matrix
    int block_size = 32;
    dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
    dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);
    // get width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA")){
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }
    // get height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA")){
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }
    // get height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB")){
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }
    // get height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB")){
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }
    // check the size of A & B
    if (dimsA.x != dimsB.y){
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n", dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);
    
    int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);

    return 0;
}