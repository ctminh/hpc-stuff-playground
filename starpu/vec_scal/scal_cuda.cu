#include "hip/hip_runtime.h"
#include <starpu.h>
#include <stdio.h>

static __global__ void vector_mult_cuda(unsigned n, float *val, float factor)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        val[i] *= factor;
}

extern "C" void scal_cuda_func(void *buffers[], void *_args)
{
    float *factor = (float *)_args;

    /* length of the vector */
    unsigned n = STARPU_VECTOR_GET_NX(buffers[0]);
    printf("[scal_cuda_func] scale the vector with factor = %f\n", *factor);

    /* load copy of the vector pointer */
    float *val = (float *)STARPU_VECTOR_GET_PTR(buffers[0]);
    unsigned threads_per_block = 64;
    unsigned nblocks = (n + threads_per_block - 1) / threads_per_block;
    vector_mult_cuda<<<nblocks, threads_per_block, 0, starpu_cuda_get_local_stream()>>> (n, val, *factor);
    printf("[scal_cuda_func] check the result: %0.2f\n", val[2]);
    
    hipStreamSynchronize(starpu_cuda_get_local_stream());
}